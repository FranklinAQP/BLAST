#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h> //printf
#include <time.h> //time, time_t, struct tm, difftime, mktime, clock_t, clock, CLOCKS_PER_SEC
#include <stdlib.h> //srand, rand
#include <algorithm> //fill_n(iterator,size,val)
#include "string.h"
#include <fstream>
#include <math.h>
#include <vector>

using namespace std;

//#define N (1024 * 1024) //longitud cadena A para procesamiento en GPU
//#define M 50 //longitud cadena B para procesamiento en GPU

#define BLOCK_SIZE 16
#define SIZE_KEY 15 //tamaño de la key
#define SUCCESS 7 //minimo valor de coincidencias a considerar en la key

long long int N, M;

//probando que un elemento output es el resultado de la suma de su input con 14 posteriores
__global__ void stencil_1d(char *A, char *B, bool *out){
	__shared__ char temp[BLOCK_SIZE + SIZE_KEY -1];
	int gindex = threadIdx.x + blockIdx.x *blockDim.x;
	int lindex = threadIdx.x;
	
	// Enviar elementos de entrada a la memoria compartida
	temp[lindex]  = A[gindex];
	if(threadIdx.x < SIZE_KEY-1) {
		temp[lindex + BLOCK_SIZE]  = A[gindex + BLOCK_SIZE];
	}

	// sincronizar  (Garantizar que todos los datos estén disponibles)
	__syncthreads();

	// Aplicar la plantilla
	int result  = 0;
	for (int i = 0 ; i < SIZE_KEY  ; ++i)
		if(*(B+i)==temp[lindex + i])		
			++result;

	// Almacena los resultados
	if(result>=SUCCESS)
		out[gindex]  = true;
}

__global__ void smith_waterman(char *A, char *B, bool *out){
	__shared__ char temp[M];
	int gindex = threadIdx.x + blockIdx.x *blockDim.x;
	int lindex = threadIdx.x;
	
	// Enviar elementos de entrada a la memoria compartida
	temp[lindex]  = A[gindex];
	if(*(out + gindex) ) {
		temp[lindex + M]  = A[gindex + M];
	}else{
		return;
	}

	// sincronizar  (Garantizar que todos los datos estén disponibles)
	__syncthreads();

	// Aplicar la plantilla

	int **matrix = (int**) malloc( (M+1) * sizeof(int *) );

    for(long long int i = 0; i < M+1; i++)
    {
	matrix[i] = (int*) malloc( (M+1) * sizeof(int));
	if(matrix[i] == NULL)
	{
	    fprintf(stderr, "No hay memoria suficiente\n");
	    return;
	}
    }

    cout<<"Iniciando matriz a 0s con \n";
    for(long long int i=0;i<=M;i++){

        for(long long int j=0;j<=M;j++){
            matrix[i][j]=0;
        }
    }

    int traceback[4];
    int I_i[M+1][M+1];
    int I_j[M+1][M+1];

    ///Comparación de cadenas con key SUCCESS
    printf ("Iniciando comparación\n");
    for (long long int k=0;k<N + SIZE_KEY -2 -M; ++k){
        if(*(out+k)==true){
            printf ("Iniciando comparación en posicion %d \n",k);
            for (long long int i=1;i<=n;i++){
                for(long long int j=0;j<=n;j++){
                    traceback[0] = matrix[i-1][j-1]+similarityScore(A+k+i-1,B+j-1);
                    traceback[1] = matrix[i-1][j]+PENALTY;
                    traceback[2] = matrix[i][j-1]+PENALTY;
                    traceback[3] = 0;
                    matrix[i][j] = findMax(&(traceback[0]),4, type_index);
                    switch(type_index){
                        case 0:
                            I_i[i][j] = i-1;
                            I_j[i][j] = j-1;
                            break;
                        case 1:
                            I_i[i][j] = i-1;
                            I_j[i][j] = j;
                            break;
                        case 2:
                            I_i[i][j] = i;
                            I_j[i][j] = j-1;
                            break;
                        case 3:
                            I_i[i][j] = i;
                            I_j[i][j] = j;
                            break;
                    }
                }
            }

            matrix_max = 0;
            int i_max=0, j_max=0;
            for(long long int i=1;i<n;i++){
                for(long long int j=1;j<n;j++){
                    if(matrix[i][j]>matrix_max){
                        matrix_max = matrix[i][j];
                        i_max=i;
                        j_max=j;
                    }
                }
            }
            //*(ind+k) = matrix_max;
            printf("Maximo escore es: %d \n",matrix_max);

        }
    }

    for(long long int i = 0; i < n+1; i++)
    {
        free (matrix[i]);
    }
    free (matrix);

	int result  = 0;
	for (int i = 0 ; i < SIZE_KEY  ; ++i)
		if(*(B+i)==temp[lindex + i])		
			++result;

	// Almacena los resultados
	if(result>=SUCCESS)
		out[gindex]  = true;
}


void fill_ints (int *x,int n) {
	fill_n(x,  n, 1);
}

void fill_ADN (int *x, int n){
	srand(time(NULL)); //Inicia random	
	for(int i=0; i<n; ++i){
		*(x+i)=rand()%4;
	}
}   

int main(void)  {
	clock_t t;	
	t = clock();
	//time_t antes, despues;	
	//double seconds;
	//time(&antes);

	string  a,b;
        char namefile_a[256];
        char namefile_b[256];
        char *A,*B; bool *out;
        char base;
        char index_a[256], index_b[256];
	
	/*Codigo para cargar archivo*/
	cout<<"\nArchivo A leido:  " << namefile_a<<"\n";
    	ifstream filea("a.txt", std::ifstream::binary);//namefile_a
    	filea.seekg(0,filea.end);///Calcula el tamano del archivo
    	long long int size_a = filea.tellg();
    	cout<<"tamaño: "<< filea.tellg()<<endl;///Imprime al tamano
    	filea.seekg(0,filea.beg);///Retorna el puntero al inicio del documento
    	A = (char*) malloc( size_a * sizeof(char) );
    	filea.read( A, size_a );
    	cout<<"Cadena: "<<endl;
    	filea.close();

	cout<<"\n\nArchivo B leido:  " << namefile_b<<"\n";
    	ifstream fileb("b.txt", std::ifstream::binary);//namefile_b
    	fileb.seekg(0,fileb.end);///Calcula el tamano del archivo
    	long long int size_b = fileb.tellg();
    	cout<<"tamano: "<<fileb.tellg()<<endl;///Imprime al tamano
    	fileb.seekg(0,fileb.beg);
	B = (char*) malloc( size_b * sizeof(char) );
    	fileb.read( B, size_b);//-z
    	fileb.close();
    	/*Codigo para cargar archivo*/
	
	cout<<"actualizando variables"<<endl;
    	N = size_a -1;
    	M = size_b -1;
	cout<<"creando variables  de almacenamiento de resultados"<<endl;	

    	long long int sizeout = (size_a -1+ SIZE_KEY) *sizeof(bool);
    	out = (bool*)malloc(sizeout); //fill_ints(out,  N + SIZE_KEY);


	//int *A, *B; bool *out; // host  copies  of a, b, c
	char *d_A, *d_B; bool *d_out; // device  copies  of a, b, c
	
	// Alloc space  for device  copies
	hipMalloc ((void **)&d_A,  N);
	hipMalloc ((void **)&d_B,  M);
	hipMalloc ((void **)&d_out, N);

	// Copy  to device
	hipMemcpy (d_A, A, N, hipMemcpyHostToDevice);
	hipMemcpy (d_B, B, M, hipMemcpyHostToDevice);
	hipMemcpy (d_out, out, N, hipMemcpyHostToDevice);

	// Launch  stencil_1d()  kernel  on  GPU
	cout<< "iniciando semillero"<<endl;
	stencil_1d<<<N/BLOCK_SIZE,BLOCK_SIZE>>>(d_A, d_B, d_out);

	// Copy  result  back  to  host
	hipMemcpy(out, d_out, N, hipMemcpyDeviceToHost);
	cout<< "terminando semillero"<<endl;
	//time(&despues);
	t = clock() - t;
	//seconds = double(despues-antes);
	//Imprimiendo
/*
	for(int i=0; i<N + SIZE_KEY -1; ++i){
		printf("valor out en pos %d es %d \n",i,*(out+i));
	}
*/
	printf ("Tiempo t: %f seconds.\n", double(t)/double(CLOCKS_PER_SEC));
	//printf ("Tiempo usado: %f seconds.\n", seconds);
	
	//Cleanup
	free(A);  free(out);
	hipFree(d_A);hipFree(d_out);
	return 0;
}
