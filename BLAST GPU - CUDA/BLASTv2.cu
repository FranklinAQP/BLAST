
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h> //printf
#include <time.h> //time, time_t, struct tm, difftime, mktime, clock_t, clock, CLOCKS_PER_SEC
#include <stdlib.h> //srand, rand
#include <algorithm> //fill_n(iterator,size,val)

using namespace std;

#define N (1024 * 1024) //longitud cadena A para procesamiento en GPU
#define M 50 //longitud cadena B para procesamiento en GPU

#define BLOCK_SIZE 16
#define SIZE_KEY 15 //tamaño de la key
#define SUCCESS 7 //minimo valor de coincidencias a considerar en la key

//probando que un elemento output es el resultado de la suma de su input con 14 posteriores
__global__ void stencil_1d(int *A, int *B, int *out){
	__shared__ int temp[BLOCK_SIZE + SIZE_KEY -1];
	int gindex = threadIdx.x + blockIdx.x *blockDim.x;
	int lindex = threadIdx.x;
	
	// Enviar elementos de entrada a la memoria compartida
	temp[lindex]  = A[gindex];
	if(threadIdx.x < SIZE_KEY-1) {
		temp[lindex + BLOCK_SIZE]  = A[gindex + BLOCK_SIZE];
	}

	// sincronizar  (Garantizar que todos los datos estén disponibles)
	__syncthreads();

	// Aplicar la plantilla
	int result  = 0;
	for (int i = 0 ; i < SIZE_KEY  ; ++i)
		if(*(B+i)==temp[lindex + i])		
			++result;

	// Almacena los resultados
	if(result>=SUCCESS)
		out[gindex]  = result;
}

void fill_ints (int *x,int n) {
	fill_n(x,  n, 1);
}

void fill_ADN (int *x, int n){
	srand(time(NULL)); //Inicia random	
	for(int i=0; i<n; ++i){
		*(x+i)=rand()%4;
	}
}   

int main(void)  {
	clock_t t;	
	t = clock();
	//time_t antes, despues;	
	//double seconds;
	//time(&antes);
	int *A, *B, *out; // host  copies  of a, b, c
	int *d_A, *d_B, *d_out; // device  copies  of a, b, c
	int sizeA  = (N + SIZE_KEY -1) *sizeof(int); 
	int sizeB  = (M) *sizeof(int); 
	

	// Alloc space  for host  copies  and  setup  values
	A  = (int *)malloc(sizeA); fill_ADN(A, N + SIZE_KEY -1);
	B  = (int *)malloc(sizeB); fill_ADN(B, M);
	out  = (int *)malloc(sizeA); fill_ints(out,  N + SIZE_KEY -1);

	// Alloc space  for device  copies
	hipMalloc ((void **)&d_A,  sizeA);
	hipMalloc ((void **)&d_B,  sizeB);
	hipMalloc ((void **)&d_out,  sizeA);

	// Copy  to device
	hipMemcpy (d_A, A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy (d_B, B, sizeB, hipMemcpyHostToDevice);
	hipMemcpy (d_out, out, sizeA, hipMemcpyHostToDevice);

	// Launch  stencil_1d()  kernel  on  GPU
	stencil_1d<<<N/BLOCK_SIZE,BLOCK_SIZE>>>(d_A, d_B, d_out);

	// Copy  result  back  to  host
	hipMemcpy(out, d_out, sizeA, hipMemcpyDeviceToHost);
	//time(&despues);
	t = clock() - t;
	//seconds = double(despues-antes);
	//Imprimiendo
/*
	for(int i=0; i<N + SIZE_KEY -1; ++i){
		printf("valor out en pos %d es %d \n",i,*(out+i));
	}
*/
	printf ("Tiempo t: %f seconds.\n", double(t)/double(CLOCKS_PER_SEC));
	//printf ("Tiempo usado: %f seconds.\n", seconds);
	
	//Cleanup
	free(A);  free(out);
	hipFree(d_A);hipFree(d_out);
	return 0;
}
